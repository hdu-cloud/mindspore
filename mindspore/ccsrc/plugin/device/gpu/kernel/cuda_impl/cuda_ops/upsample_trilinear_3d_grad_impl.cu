#include "hip/hip_runtime.h"
/**
 * Copyright 2022 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/upsample_trilinear_3d_grad_impl.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/util.cuh"

__inline__ __device__ float GetInput(const float *input, size_t index) { return input[index]; }
__inline__ __device__ float GetInput(const half *input, size_t index) { return static_cast<float>(input[index]); }

__inline__ __device__ float AreaPixelComputeScale(const size_t input_size, const size_t output_size, bool align_corners,
                                                  float scale) {
  if (align_corners) {
    if (output_size > 1) {
      return (static_cast<float>(input_size - 1)) / (static_cast<float>(output_size - 1));
    } else {
      return 0.0;
    }
  } else {
    return (scale > 0.) ? scale : static_cast<float>(input_size) / static_cast<float>(output_size);
  }
}

__inline__ __device__ float AreaPixelComputeSourceIndex(float scale, int dst_index, bool align_corners, bool cubic) {
  if (align_corners) {
    return scale * dst_index;
  } else {
    float src_idx = scale * (dst_index + 0.5) - 0.5;
    return (!cubic && src_idx < 0.0) ? 0.0 : src_idx;
  }
}

template <typename T>
__global__ void UpsampleTrilinear3DGradInitKernel(const size_t size_init, T *dx) {
  for (size_t index = blockIdx.x * blockDim.x + threadIdx.x; index < size_init; index += blockDim.x * gridDim.x) {
    dx[index] = static_cast<T>(.0);
  }
}

template <typename T>
__global__ void UpsampleTrilinear3DGrad(const T *grad, const size_t n, const size_t c, const size_t grad_d,
                                        const size_t grad_h, const size_t grad_w, const size_t grad_ncdhw,
                                        const size_t grad_cdhw, const size_t grad_dhw, const size_t grad_hw,
                                        const size_t dinput_d, const size_t dinput_h, const size_t dinput_w,
                                        const size_t dinput_cdhw, const size_t dinput_dhw, const size_t dinput_hw,
                                        const float d_scale, const float h_scale, const float w_scale,
                                        const bool align_corner, T *dinput) {
  const float rdepth = AreaPixelComputeScale(dinput_d, grad_d, align_corner, d_scale);
  const float rheight = AreaPixelComputeScale(dinput_h, grad_h, align_corner, h_scale);
  const float rwidth = AreaPixelComputeScale(dinput_w, grad_w, align_corner, w_scale);

  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < grad_ncdhw; pos += blockDim.x * gridDim.x) {
    const size_t posn = pos / grad_cdhw;
    const size_t posc = pos / grad_dhw % c;
    const size_t posd = pos / grad_hw % grad_d;
    const size_t posh = pos / grad_w % grad_h;
    const size_t posw = pos % grad_w;

    const float t1r = AreaPixelComputeSourceIndex(rdepth, posd, align_corner, false);
    const float t1 = floorf(t1r);
    const float t1p = (t1 < (dinput_d - 1)) ? 1 : 0;
    const float t1lambda = t1r - t1;
    const float t0lambda = 1.0f - t1lambda;

    const float h1r = AreaPixelComputeSourceIndex(rheight, posh, align_corner, false);
    const float h1 = floorf(h1r);
    const float h1p = (h1 < (dinput_h - 1)) ? 1 : 0;
    const float h1lambda = h1r - h1;
    const float h0lambda = 1.0f - h1lambda;

    const float w1r = AreaPixelComputeSourceIndex(rwidth, posw, align_corner, false);
    const float w1 = floorf(w1r);
    const float w1p = (w1 < (dinput_w - 1)) ? 1 : 0;
    const float w1lambda = w1r - w1;
    const float w0lambda = 1.0f - w1lambda;

    // get required indices
    const size_t p1 = posn * dinput_cdhw + posc * dinput_dhw + t1 * dinput_hw + h1 * dinput_w + w1;
    const size_t p2 = posn * dinput_cdhw + posc * dinput_dhw + t1 * dinput_hw + h1 * dinput_w + (w1 + w1p);
    const size_t p3 = posn * dinput_cdhw + posc * dinput_dhw + t1 * dinput_hw + (h1 + h1p) * dinput_w + w1;
    const size_t p4 = posn * dinput_cdhw + posc * dinput_dhw + t1 * dinput_hw + (h1 + h1p) * dinput_w + (w1 + w1p);
    const size_t p5 = posn * dinput_cdhw + posc * dinput_dhw + (t1 + t1p) * dinput_hw + h1 * dinput_w + w1;
    const size_t p6 = posn * dinput_cdhw + posc * dinput_dhw + (t1 + t1p) * dinput_hw + h1 * dinput_w + (w1 + w1p);
    const size_t p7 = posn * dinput_cdhw + posc * dinput_dhw + (t1 + t1p) * dinput_hw + (h1 + h1p) * dinput_w + w1;
    const size_t p8 =
      posn * dinput_cdhw + posc * dinput_dhw + (t1 + t1p) * dinput_hw + (h1 + h1p) * dinput_w + (w1 + w1p);

    const float d2val = GetInput(grad, pos);

    // This reduces the total number of calculations by storing repeat computations
    const float A = t0lambda * h0lambda;
    const float B = t0lambda * h1lambda;
    const float C = t1lambda * h0lambda;
    const float D = t1lambda * h1lambda;

    MsAtomicAdd(dinput + p1, static_cast<T>(A * w0lambda * d2val));
    MsAtomicAdd(dinput + p2, static_cast<T>(A * w1lambda * d2val));
    MsAtomicAdd(dinput + p3, static_cast<T>(B * w0lambda * d2val));
    MsAtomicAdd(dinput + p4, static_cast<T>(B * w1lambda * d2val));
    MsAtomicAdd(dinput + p5, static_cast<T>(C * w0lambda * d2val));
    MsAtomicAdd(dinput + p6, static_cast<T>(C * w1lambda * d2val));
    MsAtomicAdd(dinput + p7, static_cast<T>(D * w0lambda * d2val));
    MsAtomicAdd(dinput + p8, static_cast<T>(D * w1lambda * d2val));
  }
  return;
}

template <typename T>
void CalUpsampleTrilinear3DGrad(const T *grad, const size_t n, const size_t c, const size_t grad_d, const size_t grad_h,
                                const size_t grad_w, const size_t dinput_d, const size_t dinput_h,
                                const size_t dinput_w, const float d_scale, const float h_scale, const float w_scale,
                                const bool align_corner, T *dinput, hipStream_t hip_stream) {
  const size_t dinput_hw = dinput_h * dinput_w;
  const size_t dinput_dhw = dinput_d * dinput_hw;
  const size_t dinput_cdhw = c * dinput_dhw;
  const size_t dinput_ncdhw = dinput_cdhw * n;

  const size_t grad_hw = grad_h * grad_w;
  const size_t grad_dhw = grad_d * grad_hw;
  const size_t grad_cdhw = c * grad_dhw;
  const size_t grad_ncdhw = n * grad_cdhw;

  UpsampleTrilinear3DGradInitKernel<<<GET_BLOCKS(dinput_ncdhw), GET_THREADS_MAXSIZE(dinput_ncdhw), 0, hip_stream>>>(
    dinput_ncdhw, dinput);

  UpsampleTrilinear3DGrad<<<GET_BLOCKS(grad_ncdhw), GET_THREADS, 0, hip_stream>>>(
    grad, n, c, grad_d, grad_h, grad_w, grad_ncdhw, grad_cdhw, grad_dhw, grad_hw, dinput_d, dinput_h, dinput_w,
    dinput_cdhw, dinput_dhw, dinput_hw, d_scale, h_scale, w_scale, align_corner, dinput);
  return;
}

template CUDA_LIB_EXPORT void CalUpsampleTrilinear3DGrad<half>(
  const half *grad, const size_t n, const size_t c, const size_t grad_d, const size_t grad_h, const size_t grad_w,
  const size_t dinput_d, const size_t dinput_h, const size_t dinput_w, const float d_scale, const float h_scale,
  const float w_scale, const bool align_corner, half *dinput, hipStream_t hip_stream);

template CUDA_LIB_EXPORT void CalUpsampleTrilinear3DGrad<float>(
  const float *grad, const size_t n, const size_t c, const size_t grad_d, const size_t grad_h, const size_t grad_w,
  const size_t dinput_d, const size_t dinput_h, const size_t dinput_w, const float d_scale, const float h_scale,
  const float w_scale, const bool align_corner, float *dinput, hipStream_t hip_stream);

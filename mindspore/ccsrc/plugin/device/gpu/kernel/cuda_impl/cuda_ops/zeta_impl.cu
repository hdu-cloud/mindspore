#include "hip/hip_runtime.h"
/**
 * Copyright 2022 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/zeta_impl.cuh"
#include <limits>
#include "include/hip/hip_runtime.h"
#include "include/hip/hip_fp16.h"

template <typename T>
__global__ void ZetaKernel(const size_t size, const T *x, const T *dimension, T *output) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < size; pos += blockDim.x * gridDim.x) {
    double p = static_cast<double>(x[pos]);
    double q = static_cast<double>(dimension[pos]);
    const double MACHEP = static_cast<double>(1.11022302462515654042E-16);
    constexpr double zero = static_cast<double>(0.0);
    constexpr double half = static_cast<double>(0.5);
    constexpr double one = static_cast<double>(1.0);
    static const double A[] = {
      12.0,
      -720.0,
      30240.0,
      -1209600.0,
      47900160.0,
      -1.8924375803183791606e9, /*1.307674368e12/691*/
      7.47242496e10,
      -2.950130727918164224e12, /*1.067062284288e16/3617*/
      1.1646782814350067249e14, /*5.109094217170944e18/43867*/
      -4.5979787224074726105e15, /*8.028576626982912e20/174611*/
      1.8152105401943546773e17, /*1.5511210043330985984e23/854513*/
      -7.1661652561756670113e18 /*1.6938241367317436694528e27/236364091*/
    };
    int i = 0;
    double a, b, k, s, t, w;
    bool flag = false;
    if (p == one) {
      output[pos] = std::numeric_limits<T>::infinity();
      continue;
    }
    if (p < one) {
      output[pos] = std::numeric_limits<T>::quiet_NaN();
      continue;
    }
    if (q <= zero) {
      if (q == std::floor(q)) {
        output[pos] = std::numeric_limits<T>::infinity();
        continue;
      }
      if (p != std::floor(p)) {
        output[pos] = std::numeric_limits<T>::quiet_NaN();
        continue;
      }
    }
    s = std::pow(q, -p);
    a = q;
    i = 0;
    b = zero;
    while ((i < 9) || (a <= T(9.0))) {
      i += 1;
      a += one;
      b = std::pow(a, -p);
      s += b;
      if ((-MACHEP * s < b) && (b < MACHEP * s)) {
        output[pos] = static_cast<T>(s);
        flag = true;
        break;
      }
    }
    if (flag) {
      continue;
    }
    w = a;
    s += b * w / (p - one);
    s -= half * b;
    a = one;
    k = zero;
    for (int i = 0; i < 12; i++) {
      a *= p + k;
      b /= w;
      t = a * b / A[i];
      s = s + t;
      t = std::fabs(t / s);
      if (t < MACHEP) {
        output[pos] = static_cast<T>(s);
        break;
      }
      k += one;
      a *= p + k;
      b /= w;
      k += one;
    }
    output[pos] = static_cast<T>(s);
  }
  return;
}

template <typename T>
void CalZeta(const size_t size, const T *x, const T *dimension, T *output, const uint32_t &device_id,
             hipStream_t hip_stream) {
  ZetaKernel<<<CUDA_BLOCKS(device_id, size), CUDA_THREADS(device_id), 0, hip_stream>>>(size, x, dimension, output);
}

template CUDA_LIB_EXPORT void CalZeta<float>(const size_t size, const float *x, const float *dimension, float *output,
                                             const uint32_t &device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalZeta<double>(const size_t size, const double *x, const double *dimension,
                                              double *output, const uint32_t &device_id, hipStream_t hip_stream);

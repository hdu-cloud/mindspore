/**
 * Copyright 2022 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/histogram_fixed_width_impl.cuh"
#include <hipcub/hipcub.hpp>
#include "include/hip/hip_fp16.h"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/util.cuh"
#include "plugin/device/gpu/hal/device/cuda_driver.h"
#include "plugin/device/gpu/hal/device/gpu_memory_allocator.h"

template <typename T>
void HistogramFixedWidthKernel(int num_samples, const T *d_samples, const double *d_levels,
                               int32_t *d_histogram, int64_t num_levels, hipStream_t hip_stream) {
  void *d_temp_storage = nullptr;
  size_t temp_storage_bytes = 0;
  (void)hipcub::DeviceHistogram::HistogramRange(nullptr, temp_storage_bytes, d_samples, d_histogram, num_levels, d_levels,
                                             num_samples, hip_stream);
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  (void)hipcub::DeviceHistogram::HistogramRange(d_temp_storage, temp_storage_bytes, d_samples, d_histogram, num_levels,
                                             d_levels, num_samples, hip_stream);
  (void)hipFree(d_temp_storage);
  return;
}

template <typename T>
void CalHistogramFixedWidth(int num_samples, const T *d_samples, const double *d_levels, int32_t *d_histogram,
                            int64_t num_levels, hipStream_t hip_stream) {
  HistogramFixedWidthKernel(num_samples, d_samples, d_levels, d_histogram, num_levels, hip_stream);
  return;
}

template CUDA_LIB_EXPORT void CalHistogramFixedWidth<int32_t>(int num_samples, const int32_t *d_samples,
                                                              const double *d_levels, int32_t *d_histogram,
                                                              int64_t num_levels, hipStream_t hip_stream);

template CUDA_LIB_EXPORT void CalHistogramFixedWidth<double>(int num_samples, const double *d_samples,
                                                             const double *d_levels, int32_t *d_histogram,
                                                             int64_t num_levels, hipStream_t hip_stream);

template CUDA_LIB_EXPORT void CalHistogramFixedWidth<float>(int num_samples, const float *d_samples,
                                                            const double *d_levels, int32_t *d_histogram,
                                                            int64_t num_levels, hipStream_t hip_stream);

template CUDA_LIB_EXPORT void CalHistogramFixedWidth<half>(int num_samples, const half *d_samples,
                                                           const double *d_levels, int32_t *d_histogram,
                                                           int64_t num_levels, hipStream_t hip_stream);

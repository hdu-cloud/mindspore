#include "hip/hip_runtime.h"
/**
 * Copyright 2022 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "mvlgamma_impl.cuh"
#ifdef _WIN32
// for M_PI
#define _USE_MATH_DEFINES
#include <math.h>
#endif

template <typename T>
__global__ void Valid(bool *valid, const size_t size, const T *input, const int p) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < size; pos += blockDim.x * gridDim.x) {
    if (input[pos] <= (0.5 * (p - 1))) {
      *valid = false;
      return;
    }
  }
  return;
}

template <typename T>
__global__ void Mvlgamma(const size_t size, const T *input, const int p, T *output) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < size; pos += blockDim.x * gridDim.x) {
    T temp = 0;
    for (int i = 1; i <= p; i++) {
      temp += lgamma(input[pos] - static_cast<T>((i - 1) * 0.5));
    }
    output[pos] = temp + static_cast<T>(p * (p - 1) * 0.25 * log(M_PI));
  }
  return;
}

template <typename T>
void CalMvlgamma(bool *valid, const size_t size, const T *input, const int p, T *output, const uint32_t &device_id,
                 hipStream_t hip_stream) {
  Valid<<<CUDA_BLOCKS(device_id, size), CUDA_THREADS(device_id), 0, hip_stream>>>(valid, size, input, p);
  bool host_valid = true;
  hipMemcpyFromSymbol(&host_valid, HIP_SYMBOL(valid), sizeof(bool));
  if (!host_valid) {
    return;
  }
  Mvlgamma<<<CUDA_BLOCKS(device_id, size), CUDA_THREADS(device_id), 0, hip_stream>>>(size, input, p, output);
  return;
}

template
CUDA_LIB_EXPORT void CalMvlgamma<float>(bool *valid, const size_t size, const float *input, const int p,
                                        float *output, const uint32_t &device_id, hipStream_t hip_stream);
template
CUDA_LIB_EXPORT void CalMvlgamma<double>(bool *valid, const size_t size, const double *input, const int p,
                                         double *output, const uint32_t &device_id, hipStream_t hip_stream);

#include "hip/hip_runtime.h"
/**
 * Copyright 2022 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "inplace_update_impl.cuh"

template <typename T>
__global__ void InplaceUpdate(const size_t size, const T *input_v, T *output, const int64_t *indices,
                              const int64_t band_size) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < size; pos += blockDim.x * gridDim.x) {
    int v_row = pos / band_size;
    int x_row = indices[v_row];
    int offset = pos % band_size;
    int x_offset = x_row * band_size;
    output[x_offset + offset] = input_v[pos];
  }
  return;
}

template <typename T>
void CalInplaceUpdate(const size_t size_v, const T *input_v, T *output, const int64_t *indices, const int64_t band_size,
                      const uint32_t &device_id, hipStream_t hip_stream) {
  InplaceUpdate<<<CUDA_BLOCKS(device_id, size_v), CUDA_THREADS(device_id), 0, hip_stream>>>(size_v, input_v, output,
                                                                                           indices, band_size);
  return;
}

template CUDA_LIB_EXPORT void CalInplaceUpdate<half>(const size_t size_v, const half *input_v, half *output,
                                                     const int64_t *indices, const int64_t band_size,
                                                     const uint32_t &device_id, hipStream_t hip_stream);

template CUDA_LIB_EXPORT void CalInplaceUpdate<float>(const size_t size_v, const float *input_v, float *output,
                                                      const int64_t *indices, const int64_t band_size,
                                                      const uint32_t &device_id, hipStream_t hip_stream);

template CUDA_LIB_EXPORT void CalInplaceUpdate<double>(const size_t size_v, const double *input_v, double *output,
                                                       const int64_t *indices, const int64_t band_size,
                                                       const uint32_t &device_id, hipStream_t hip_stream);

template CUDA_LIB_EXPORT void CalInplaceUpdate<int>(const size_t size_v, const int *input_v, int *output,
                                                    const int64_t *indices, const int64_t band_size,
                                                    const uint32_t &device_id, hipStream_t hip_stream);

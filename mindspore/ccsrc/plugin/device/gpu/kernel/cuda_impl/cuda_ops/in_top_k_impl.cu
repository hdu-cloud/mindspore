#include "hip/hip_runtime.h"
/**
 * Copyright 2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "in_top_k_impl.cuh"
#include <hip/hip_runtime.h>
#include "include/hip/hip_fp16.h"

__device__ __forceinline__ bool Isfinite(half x) {
  return isfinite(static_cast<float>(x));
}

template <typename T>
__device__ __forceinline__ bool Isfinite(T x) {
  return isfinite(x);
}

template <typename T, typename S>
__global__ void InTopK(const T *predictions, const S *targets, bool *output, const T *top_k_output, size_t batch_size,
                       size_t class_id_count, int64_t k) {
  size_t gt_id = blockIdx.x * blockDim.x + threadIdx.x;
  for (; gt_id < batch_size; gt_id += blockDim.x * gridDim.x) {
    S target_index = targets[gt_id];
    bool is_invalid = (static_cast<size_t>(target_index) >= class_id_count);
    if (!is_invalid) {
      T predicted_value = predictions[gt_id * class_id_count + target_index];
      T top_k_smallest_value = top_k_output[gt_id * k + k - 1];
      is_invalid = is_invalid || !Isfinite(predicted_value);
      output[gt_id] = is_invalid ? false : predicted_value >= top_k_smallest_value;
    } else {
      output[gt_id] = false;
    }
  }
}

template <typename T, typename S>
void CalInTopK(const T *predictions, const S *targets, bool *output, const T *top_k_output, size_t batch_size,
               size_t class_id_count, int64_t k, hipStream_t hip_stream) {
  InTopK<<<GET_BLOCKS(class_id_count), GET_THREADS, 0, hip_stream>>>(predictions, targets, output, top_k_output,
                                                                      batch_size, class_id_count, k);
}

template CUDA_LIB_EXPORT void CalInTopK<half, int32_t>(const half *predictions, const int32_t *targets, bool *output,
                                                       const half *top_k_output, size_t batch_size,
                                                       size_t class_id_count, int64_t k, hipStream_t hip_stream);

template CUDA_LIB_EXPORT void CalInTopK<float, int32_t>(const float *predictions, const int32_t *targets, bool *output,
                                                        const float *top_k_output, size_t batch_size,
                                                        size_t class_id_count, int64_t k, hipStream_t hip_stream);

template CUDA_LIB_EXPORT void CalInTopK<half, int64_t>(const half *predictions, const int64_t *targets, bool *output,
                                                       const half *top_k_output, size_t batch_size,
                                                       size_t class_id_count, int64_t k, hipStream_t hip_stream);

template CUDA_LIB_EXPORT void CalInTopK<float, int64_t>(const float *predictions, const int64_t *targets, bool *output,
                                                        const float *top_k_output, size_t batch_size,
                                                        size_t class_id_count, int64_t k, hipStream_t hip_stream);

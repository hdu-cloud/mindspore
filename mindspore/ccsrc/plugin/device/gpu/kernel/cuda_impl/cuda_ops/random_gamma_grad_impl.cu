#include "hip/hip_runtime.h"
/**
 * Copyright 2022 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "random_gamma_grad_impl.cuh"
#include <limits>
#include <algorithm>

template <typename T>
__device__ inline T polevl(const T x, const T A[], size_t len) {
  T result = 0;
  for (size_t i = 0; i <= len; i++) {
    result = result * x + A[i];
  }
  return result;
}

template <typename T>
__device__ inline T Digamma(T input) {
  const T PI = static_cast<T>(3.14159265358979323846);
  const T PSI_10 = static_cast<T>(2.25175258906672110764);

  T output = 0;
  if (input < 0) {
    if (input == trunc(input)) {
      return std::numeric_limits<T>::quiet_NaN();
    }
    output = static_cast<T>(-PI / tan(PI * input));
    input = 1 - input;
  }
  while (input < 10) {
    output -= 1 / input;
    input += 1;
  }
  if (input == 10) {
    return static_cast<T>(output + PSI_10);
  }

  const T A[] = {8.33333333333333333333E-2,  -2.10927960927960927961E-2, 7.57575757575757575758E-3,
                 -4.16666666666666666667E-3, 3.96825396825396825397E-3,  -8.33333333333333333333E-3,
                 8.33333333333333333333E-2};
  T y = 0;
  if (input < 1.0e17) {
    T z = 1.0 / (input * input);
    y = z * polevl(z, A, 6);
  }

  return static_cast<T>(log(input) - 0.5 / input - y + output);
}

template <typename dtype>
__device__ inline dtype IgammaSeries(dtype aSingle, dtype xSingle) {
  dtype enabled = 1, ans = 1, c_muti = 1;
  dtype r_plus_one = aSingle;
  dtype dc_da = 0, dans_da = 0, dlogax_da = 0;
  while (enabled != 0) {
    r_plus_one += 1;
    dc_da = dc_da * (xSingle / r_plus_one) + (-1 * c_muti * xSingle) / (r_plus_one * r_plus_one);
    dans_da += dc_da;
    c_muti *= (xSingle / r_plus_one);
    ans += c_muti;
    enabled = enabled && (fabs(dc_da / dans_da) > std::numeric_limits<dtype>::epsilon());
  }
  dlogax_da = log(xSingle) - Digamma(aSingle + 1);
  return static_cast<dtype>(-(dans_da + ans * dlogax_da) * xSingle / aSingle);
}

template <typename Intype>
__device__ inline Intype IgammacContinuedFraction(Intype aSingle, Intype xSingle) {
  Intype y_plus_one = 1 - aSingle;
  Intype z_plus_two = xSingle + y_plus_one + 1;
  Intype c_plus_one = 0, dpkm2_da = 0, dqkm2_da = 0, dpkm1_da = 0;
  Intype pkm2 = 1;
  Intype qkm2 = xSingle;
  Intype pkm1 = xSingle + 1;
  Intype qkm1 = z_plus_two * xSingle;
  Intype ans = pkm1 / qkm1;
  Intype dqkm1_da = - xSingle;
  Intype dans_da = (dpkm1_da - ans * dqkm1_da) / qkm1;
  for (size_t i = 0; i < 2000; i++) {
    c_plus_one += 1;
    y_plus_one += 1;
    z_plus_two += 2;
    Intype y_muti_c = y_plus_one * c_plus_one;
    Intype pk = pkm1 * z_plus_two - pkm2 * y_muti_c;
    Intype qk = qkm1 * z_plus_two - qkm2 * y_muti_c;
    Intype dpk_da = dpkm1_da * z_plus_two - pkm1 - dpkm2_da * y_muti_c + pkm2 * c_plus_one;
    Intype dqk_da = dqkm1_da * z_plus_two - qkm1 - dqkm2_da * y_muti_c + qkm2 * c_plus_one;
    if (qk != 0) {
      ans = pk / qk;
      Intype dans_da_new = dans_da;
      dans_da = (dpk_da - ans * dqk_da) / qk;
      if (fabs(dans_da - dans_da_new) < std::numeric_limits<Intype>::epsilon()) {
        break;
      }
    }
    pkm2 = pkm1;
    pkm1 = pk;
    qkm2 = qkm1;
    qkm1 = qk;

    dpkm2_da = dpkm1_da;
    dqkm2_da = dqkm1_da;
    dpkm1_da = dpk_da;
    dqkm1_da = dqk_da;
    bool rescale = fabs(pk) > (1 / std::numeric_limits<Intype>::epsilon());
    pkm2 = rescale ? pkm2 * std::numeric_limits<Intype>::epsilon() : pkm2;
    pkm1 = rescale ? pkm1 * std::numeric_limits<Intype>::epsilon() : pkm1;
    qkm2 = rescale ? qkm2 * std::numeric_limits<Intype>::epsilon() : qkm2;
    qkm1 = rescale ? qkm1 * std::numeric_limits<Intype>::epsilon() : qkm1;
    dpkm2_da = rescale ? dpkm2_da * std::numeric_limits<Intype>::epsilon() : dpkm2_da;
    dqkm2_da = rescale ? dqkm2_da * std::numeric_limits<Intype>::epsilon() : dqkm2_da;
    dpkm1_da = rescale ? dpkm1_da * std::numeric_limits<Intype>::epsilon() : dpkm1_da;
    dqkm1_da = rescale ? dqkm1_da * std::numeric_limits<Intype>::epsilon() : dqkm1_da;
  }
  Intype dlogax_da = log(xSingle) - Digamma(aSingle);
  return static_cast<Intype>((dans_da + ans * dlogax_da) * xSingle);
}

template <typename type>
__device__ inline type GammaSingle(type aSingle, type xSingle) {
  type ax = aSingle * log(xSingle) - xSingle - lgamma(aSingle);
  bool is_nonzero = (xSingle < 0) || (aSingle <= 0);
  bool is_nan = isnan(aSingle) || isnan(xSingle);
  bool underflow = ax < -log(std::numeric_limits<type>::max());
  bool x_is_zero = xSingle == 0;
  if (is_nan || is_nonzero || underflow || x_is_zero) {
    return std::numeric_limits<type>::quiet_NaN();
  }
  bool use_igammac = (xSingle > 1) && (xSingle > aSingle);
  type result;
  if (use_igammac) {
    result = IgammacContinuedFraction<type>(aSingle, xSingle);
  } else {
    result = IgammaSeries<type>(aSingle, xSingle);
  }
  return result;
}

template <typename T>
__global__ void RandomGammaGradKernel(const T *alpha, const T *sample, T *output, int elements) {
  for (int pos = blockIdx.x * blockDim.x + threadIdx.x; pos < elements; pos += blockDim.x * gridDim.x) {
    output[pos] = GammaSingle<T>(alpha[pos], sample[pos]);
  }
}

__device__ __forceinline__ size_t Index(const size_t &index, const size_t &dim) { return dim == 1 ? 0 : index; }

template <typename T>
__global__ void BroadcastRandomGammaGradKernel(size_t i0, size_t i1, size_t i2, size_t i3, size_t i4, size_t i5,
                                               size_t i6, size_t j0, size_t j1, size_t j2, size_t j3, size_t j4,
                                               size_t j5, size_t j6, size_t o0, size_t o1, size_t o2, size_t o3,
                                               size_t o4, size_t o5, size_t o6, const T *alpha, const T *sample,
                                               T *output) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < o0 * o1 * o2 * o3 * o4 * o5 * o6;
       pos += blockDim.x * gridDim.x) {
    size_t i = pos / (o1 * o2 * o3 * o4 * o5 * o6) % o0;
    size_t j = pos / (o2 * o3 * o4 * o5 * o6) % o1;
    size_t k = pos / (o3 * o4 * o5 * o6) % o2;
    size_t l = pos / (o4 * o5 * o6) % o3;
    size_t m = pos / (o5 * o6) % o4;
    size_t n = pos / o6 % o5;
    size_t o = pos % o6;

    size_t inputx_idx = Index(i, i0) * i1 * i2 * i3 * i4 * i5 * i6;
    inputx_idx += Index(j, i1) * i2 * i3 * i4 * i5 * i6;
    inputx_idx += Index(k, i2) * i3 * i4 * i5 * i6;
    inputx_idx += Index(l, i3) * i4 * i5 * i6;
    inputx_idx += Index(m, i4) * i5 * i6;
    inputx_idx += Index(n, i5) * i6;
    inputx_idx += Index(o, i6);

    size_t inputy_idx = Index(i, j0) * j1 * j2 * j3 * j4 * j5 * j6;
    inputy_idx += Index(j, j1) * j2 * j3 * j4 * j5 * j6;
    inputy_idx += Index(k, j2) * j3 * j4 * j5 * j6;
    inputy_idx += Index(l, j3) * j4 * j5 * j6;
    inputy_idx += Index(m, j4) * j5 * j6;
    inputy_idx += Index(n, j5) * j6;
    inputy_idx += Index(o, j6);
    output[pos] = GammaSingle<T>(alpha[inputx_idx], sample[inputy_idx]);
  }
}

template <typename T>
void CalRandomGammaGrad(const T *alpha, const T *sample, T *output, int elements, const uint32_t &device_id,
                        hipStream_t hip_stream) {
  int thread_num = 1024 < elements ? 1024 : elements;
  hipDeviceProp_t prop;
  (void)hipGetDeviceProperties(&prop, device_id);
  int max_blocks = prop.multiProcessorCount;
  int block_num = std::min(static_cast<int>(((elements - 1) / thread_num) + 1), max_blocks);
  RandomGammaGradKernel<<<block_num, thread_num, 0, hip_stream>>>(alpha, sample,
    output, elements);
}

template <typename T>
void BroadcastRandomGammaGrad(const std::vector<size_t> &alpha_shape, const std::vector<size_t> &sample_shape,
                              const std::vector<size_t> &output_shape, const T *alpha, const T *sample, T *output,
                              const uint32_t &device_id, hipStream_t hip_stream) {
  size_t size = 1;
  for (auto d : output_shape) {
    size *= d;
  }
  int thread_num = 1024 < size ? 1024 : size;
  hipDeviceProp_t prop;
  (void)hipGetDeviceProperties(&prop, device_id);
  int max_blocks = prop.multiProcessorCount;
  int block_num = std::min(static_cast<int>(((size - 1) / thread_num) + 1), max_blocks);
  BroadcastRandomGammaGradKernel<<<block_num, thread_num, 0, hip_stream>>>(
    alpha_shape[0], alpha_shape[1], alpha_shape[2], alpha_shape[3], alpha_shape[4], alpha_shape[5], alpha_shape[6],
    sample_shape[0], sample_shape[1], sample_shape[2], sample_shape[3], sample_shape[4], sample_shape[5],
    sample_shape[6], output_shape[0], output_shape[1], output_shape[2], output_shape[3], output_shape[4],
    output_shape[5], output_shape[6], alpha, sample, output);
}

template CUDA_LIB_EXPORT void CalRandomGammaGrad<double>(const double *alpha, const double *sample, double *output,
                                                         int elements, const uint32_t &device_id,
                                                         hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalRandomGammaGrad<float>(const float *alpha, const float *sample, float *output,
                                                        int elements, const uint32_t &device_id,
                                                        hipStream_t hip_stream);

template CUDA_LIB_EXPORT void BroadcastRandomGammaGrad<double>(const std::vector<size_t> &, const std::vector<size_t> &,
                                                               const std::vector<size_t> &, const double *,
                                                               const double *, double *, const uint32_t &,
                                                               hipStream_t hip_stream);
template CUDA_LIB_EXPORT void BroadcastRandomGammaGrad<float>(const std::vector<size_t> &, const std::vector<size_t> &,
                                                              const std::vector<size_t> &, const float *, const float *,
                                                              float *, const uint32_t &, hipStream_t hip_stream);

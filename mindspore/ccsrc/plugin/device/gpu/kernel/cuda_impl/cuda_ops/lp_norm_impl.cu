#include "hip/hip_runtime.h"
/**
 * Copyright 2022 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/lp_norm_impl.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/util.cuh"

template <typename T>
__inline__ __device__ void LpNormCoreOp(const T *input, size_t input_index, float *output, size_t output_index,
                                        float p) {
  T abs_value = input[input_index] >= static_cast<T>(0) ? input[input_index] : -input[input_index];
  // We do parallel LpNorm by input elements. So multiple input data will be reduce sum to output, which causes data
  // competition.
  (void)MsAtomicAdd(output + output_index, pow(abs_value, p));
}

template <typename T>
__global__ void LpCalKernel(const T *input, const size_t *input_shape, size_t input_shape_length, size_t input_elements,
                            const size_t *output_axis, const size_t *output_stride, size_t output_shape_length, float p,
                            float eps, float *middle_output) {
  for (size_t index = blockIdx.x * blockDim.x + threadIdx.x; index < (input_elements);
       index += blockDim.x * gridDim.x) {
    size_t flatten_index = index;
    size_t output_index = 0;
    for (int i = static_cast<int>(input_shape_length - 1); i >= 0; --i) {
      for (int j = static_cast<int>(output_shape_length - 1); j >= 0; --j) {
        // 1. Calculate coordinate by input shape.
        size_t coordinate = flatten_index % input_shape[i];
        // 2. Map input coordinate to output coordinate by axis.
        if (i == output_axis[j]) {
          // 3. Calculate output index by output coordinate.
          output_index += coordinate * output_stride[j];
          break;
        }
      }
      flatten_index = flatten_index / input_shape[i];
    }
    LpNormCoreOp(input, index, middle_output, output_index, p);
  }
}

template <typename T>
__global__ void NormCalKernel(T *output, size_t output_elements, float p, float eps) {
  for (size_t index = blockIdx.x * blockDim.x + threadIdx.x; index < (output_elements);
       index += blockDim.x * gridDim.x) {
    output[index] = max(pow(output[index], 1 / p), eps);
  }
}

template <typename T>
__global__ void NormCalHighPrecisionKernel(const float *middle_output, T *output, size_t output_elements, float p,
                                           float eps) {
  for (size_t index = blockIdx.x * blockDim.x + threadIdx.x; index < (output_elements);
       index += blockDim.x * gridDim.x) {
    output[index] = pow(middle_output[index], 1 / p);
  }
}

template <>
void CalLpNorm<float>(const float *input, const size_t *input_shape, size_t input_shape_length,
                      size_t input_elements, const size_t *output_axis, const size_t *output_stride,
                      size_t output_shape_length, size_t output_elements, float p, float eps,
                      float *middle_output, float *output, const uint32_t &device_id,
                      hipStream_t hip_stream) {
  LpCalKernel<<<CUDA_BLOCKS(device_id, input_elements), CUDA_THREADS(device_id), 0, hip_stream>>>(
    input, input_shape, input_shape_length, input_elements, output_axis, output_stride, output_shape_length, p, eps,
    output);
  NormCalKernel<<<CUDA_BLOCKS(device_id, output_elements), CUDA_THREADS(device_id), 0, hip_stream>>>(
    output, output_elements, p, eps);
}

template <>
void CalLpNorm<half>(const half *input, const size_t *input_shape, size_t input_shape_length,
                     size_t input_elements, const size_t *output_axis, const size_t *output_stride,
                     size_t output_shape_length, size_t output_elements, float p, float eps,
                     float *middle_output, half *output, const uint32_t &device_id,
                     hipStream_t hip_stream) {
  LpCalKernel<<<CUDA_BLOCKS(device_id, input_elements), CUDA_THREADS(device_id), 0, hip_stream>>>(
    input, input_shape, input_shape_length, input_elements, output_axis, output_stride, output_shape_length, p, eps,
    middle_output);
  NormCalHighPrecisionKernel<<<CUDA_BLOCKS(device_id, output_elements), CUDA_THREADS(device_id), 0, hip_stream>>>(
    middle_output, output, output_elements, p, eps);
}

template CUDA_LIB_EXPORT
void CalLpNorm<float>(const float *input, const size_t *input_shape, size_t input_shape_length,
                      size_t input_elements, const size_t *output_axis, const size_t *output_stride,
                      size_t output_shape_length, size_t output_elements, float p, float eps,
                      float *middle_output, float *output, const uint32_t &device_id,
                      hipStream_t hip_stream);

template CUDA_LIB_EXPORT
void CalLpNorm<half>(const half *input, const size_t *input_shape, size_t input_shape_length,
                     size_t input_elements, const size_t *output_axis, const size_t *output_stride,
                     size_t output_shape_length, size_t output_elements, float p, float eps,
                     float *middle_output, half *output, const uint32_t &device_id,
                     hipStream_t hip_stream);

#include "hip/hip_runtime.h"
/**
 * Copyright 2020 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <iostream>
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/gather_grad.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/complex.h"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/util.cuh"

template <typename T>
using Complex = mindspore::utils::Complex<T>;

template <typename T, typename S>
__global__ void GatherGradKernel(const size_t num, const T *index, const S *grad, S *output,
                                 const size_t dim_before_axis, const size_t dim_at_axis_index,
                                 const size_t dim_at_axis_output, const size_t dim_after_axis) {
  size_t i, k;

  for (size_t id = blockIdx.x * blockDim.x + threadIdx.x; id < num; id += blockDim.x * gridDim.x) {
    i = id / (dim_at_axis_index * dim_after_axis);
    k = id % dim_after_axis;

    T j = index[id];
    if (j < 0) {
      j += static_cast<T>(dim_at_axis_output);
    }
    CUDA_KERNEL_ASSERT(j >= 0);
    size_t j_read = static_cast<size_t>(j);
    CUDA_KERNEL_ASSERT(j_read < dim_at_axis_output);
    size_t read_id = i * dim_at_axis_output * dim_after_axis + j_read * dim_after_axis + k;
    MsAtomicAdd(output + read_id, grad[id]);
  }
  return;
}

template <typename T, typename S>
hipError_t GatherGrad(const T *index, const S *grad, S *output, const size_t dim_before_axis,
                       const size_t dim_at_axis_index, const size_t dim_at_axis_output, const size_t dim_after_axis,
                       hipStream_t stream) {
  size_t size = dim_before_axis * dim_at_axis_index * dim_after_axis;
  GatherGradKernel<<<GET_BLOCKS(size), GET_THREADS, 0, stream>>>(size, index, grad, output, dim_before_axis,
                                                                 dim_at_axis_index, dim_at_axis_output, dim_after_axis);
  return GetCudaStatus();
}

template CUDA_LIB_EXPORT hipError_t GatherGrad<int, Complex<double>>(
  const int *index, const Complex<double> *grad, Complex<double> *output, const size_t dim_before_axis,
  const size_t dim_at_axis_index, const size_t dim_at_axis_output, const size_t dim_after_axis, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t GatherGrad<int64_t, Complex<double>>(
  const int64_t *index, const Complex<double> *grad, Complex<double> *output, const size_t dim_before_axis,
  const size_t dim_at_axis_index, const size_t dim_at_axis_output, const size_t dim_after_axis, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t GatherGrad<int, Complex<float>>(
  const int *index, const Complex<float> *grad, Complex<float> *output, const size_t dim_before_axis,
  const size_t dim_at_axis_index, const size_t dim_at_axis_output, const size_t dim_after_axis, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t GatherGrad<int64_t, Complex<float>>(
  const int64_t *index, const Complex<float> *grad, Complex<float> *output, const size_t dim_before_axis,
  const size_t dim_at_axis_index, const size_t dim_at_axis_output, const size_t dim_after_axis, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t GatherGrad<int, double>(const int *index, const double *grad, double *output,
                                                             const size_t dim_before_axis,
                                                             const size_t dim_at_axis_index,
                                                             const size_t dim_at_axis_output,
                                                             const size_t dim_after_axis, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t GatherGrad<int64_t, double>(const int64_t *index, const double *grad,
                                                                 double *output, const size_t dim_before_axis,
                                                                 const size_t dim_at_axis_index,
                                                                 const size_t dim_at_axis_output,
                                                                 const size_t dim_after_axis, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t GatherGrad<int, float>(const int *index, const float *grad, float *output,
                                                            const size_t dim_before_axis,
                                                            const size_t dim_at_axis_index,
                                                            const size_t dim_at_axis_output,
                                                            const size_t dim_after_axis, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t GatherGrad<int64_t, float>(const int64_t *index, const float *grad, float *output,
                                                                const size_t dim_before_axis,
                                                                const size_t dim_at_axis_index,
                                                                const size_t dim_at_axis_output,
                                                                const size_t dim_after_axis, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t GatherGrad<int, half>(const int *index, const half *grad, half *output,
                                                           const size_t dim_before_axis, const size_t dim_at_axis_index,
                                                           const size_t dim_at_axis_output, const size_t dim_after_axis,
                                                           hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t GatherGrad<int64_t, half>(const int64_t *index, const half *grad, half *output,
                                                               const size_t dim_before_axis,
                                                               const size_t dim_at_axis_index,
                                                               const size_t dim_at_axis_output,
                                                               const size_t dim_after_axis, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t GatherGrad<int, int>(const int *index, const int *grad, int *output,
                                                          const size_t dim_before_axis, const size_t dim_at_axis_index,
                                                          const size_t dim_at_axis_output, const size_t dim_after_axis,
                                                          hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t GatherGrad<int64_t, int>(const int64_t *index, const int *grad, int *output,
                                                              const size_t dim_before_axis,
                                                              const size_t dim_at_axis_index,
                                                              const size_t dim_at_axis_output,
                                                              const size_t dim_after_axis, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t GatherGrad<int, int8_t>(const int *index, const int8_t *grad, int8_t *output,
                                                             const size_t dim_before_axis,
                                                             const size_t dim_at_axis_index,
                                                             const size_t dim_at_axis_output,
                                                             const size_t dim_after_axis, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t GatherGrad<int64_t, int8_t>(const int64_t *index, const int8_t *grad,
                                                                 int8_t *output, const size_t dim_before_axis,
                                                                 const size_t dim_at_axis_index,
                                                                 const size_t dim_at_axis_output,
                                                                 const size_t dim_after_axis, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t GatherGrad<int, int16_t>(const int *index, const int16_t *grad, int16_t *output,
                                                              const size_t dim_before_axis,
                                                              const size_t dim_at_axis_index,
                                                              const size_t dim_at_axis_output,
                                                              const size_t dim_after_axis, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t GatherGrad<int64_t, int16_t>(const int64_t *index, const int16_t *grad,
                                                                  int16_t *output, const size_t dim_before_axis,
                                                                  const size_t dim_at_axis_index,
                                                                  const size_t dim_at_axis_output,
                                                                  const size_t dim_after_axis, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t GatherGrad<int, int64_t>(const int *index, const int64_t *grad, int64_t *output,
                                                              const size_t dim_before_axis,
                                                              const size_t dim_at_axis_index,
                                                              const size_t dim_at_axis_output,
                                                              const size_t dim_after_axis, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t GatherGrad<int64_t, int64_t>(const int64_t *index, const int64_t *grad,
                                                                  int64_t *output, const size_t dim_before_axis,
                                                                  const size_t dim_at_axis_index,
                                                                  const size_t dim_at_axis_output,
                                                                  const size_t dim_after_axis, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t GatherGrad<int, unsigned char>(const int *index, const unsigned char *grad,
                                                                    unsigned char *output, const size_t dim_before_axis,
                                                                    const size_t dim_at_axis_index,
                                                                    const size_t dim_at_axis_output,
                                                                    const size_t dim_after_axis, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t GatherGrad<int64_t, unsigned char>(
  const int64_t *index, const unsigned char *grad, unsigned char *output, const size_t dim_before_axis,
  const size_t dim_at_axis_index, const size_t dim_at_axis_output, const size_t dim_after_axis, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t GatherGrad<int, uint64_t>(const int *index, const uint64_t *grad, uint64_t *output,
                                                               const size_t dim_before_axis,
                                                               const size_t dim_at_axis_index,
                                                               const size_t dim_at_axis_output,
                                                               const size_t dim_after_axis, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t GatherGrad<int64_t, uint64_t>(const int64_t *index, const uint64_t *grad,
                                                                   uint64_t *output, const size_t dim_before_axis,
                                                                   const size_t dim_at_axis_index,
                                                                   const size_t dim_at_axis_output,
                                                                   const size_t dim_after_axis, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t GatherGrad<int, uint32_t>(const int *index, const uint32_t *grad, uint32_t *output,
                                                               const size_t dim_before_axis,
                                                               const size_t dim_at_axis_index,
                                                               const size_t dim_at_axis_output,
                                                               const size_t dim_after_axis, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t GatherGrad<int64_t, uint32_t>(const int64_t *index, const uint32_t *grad,
                                                                   uint32_t *output, const size_t dim_before_axis,
                                                                   const size_t dim_at_axis_index,
                                                                   const size_t dim_at_axis_output,
                                                                   const size_t dim_after_axis, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t GatherGrad<int, uint16_t>(const int *index, const uint16_t *grad, uint16_t *output,
                                                               const size_t dim_before_axis,
                                                               const size_t dim_at_axis_index,
                                                               const size_t dim_at_axis_output,
                                                               const size_t dim_after_axis, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t GatherGrad<int64_t, uint16_t>(const int64_t *index, const uint16_t *grad,
                                                                   uint16_t *output, const size_t dim_before_axis,
                                                                   const size_t dim_at_axis_index,
                                                                   const size_t dim_at_axis_output,
                                                                   const size_t dim_after_axis, hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t GatherGrad<int, bool>(const int *index, const bool *grad, bool *output,
                                                           const size_t dim_before_axis, const size_t dim_at_axis_index,
                                                           const size_t dim_at_axis_output, const size_t dim_after_axis,
                                                           hipStream_t stream);
template CUDA_LIB_EXPORT hipError_t GatherGrad<int64_t, bool>(const int64_t *index, const bool *grad, bool *output,
                                                               const size_t dim_before_axis,
                                                               const size_t dim_at_axis_index,
                                                               const size_t dim_at_axis_output,
                                                               const size_t dim_after_axis, hipStream_t stream);
